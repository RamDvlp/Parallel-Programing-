#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


// Initialize the temporary array
__global__ void initTemp(int* arr) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int i;
    int offset = id*256;
  
    for (i = offset;   i < offset+256;   i++)   // Each of 200 threads initialize 256 members in temp
        arr[i] = 0;  
  
}

// Kernel to create the temporary array
__global__ void fillTemp(int* d_A, int *d_temp, int size, int range) {
  int id = threadIdx.x + blockIdx.x * blockDim.x; //0-199
  int chunk = size/200; //75000 / 200 = 375
  int offset_data = id * chunk;  // Start of the part of data for this thread
  int offset_temp = id *range; // Start of the part of temp for this thread
  int i, index;

  // Jump to the place in data and update the proper part of the temp
  for (i = 0;   i < chunk;  i++) {
    index = d_A[offset_data + i];
    d_temp[offset_temp + index]++;
  }
}

// Unify all values in the temp
__global__ void unify(int *d_temp, int *d_out) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for(int i = tid ;i< 256; i+=200 ) { // Threads 0-55 will make "double shift"
        int result = 0;
        for(int bin = 0; bin < 200; bin++){
            result += d_temp[i + bin*256];
        }
        d_out[i] = result;
    }

}



int computeOnGPU(int *data, int numElements,int* cudaOut, int cudaOutSize) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(int); // N/4 data
    size_t outSize = cudaOutSize * sizeof(int); // 256 values
    size_t tempSize = cudaOutSize * NUM_OF_BLOCKS * NUM_OF_THREADS * sizeof(int); //256 * 10 * 20 = 51200 temp array

    // Allocate memory on GPU to copy the data from the host
    int *d_A, *d_out, *d_temp;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate data device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_out, outSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate result device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_temp, tempSize);
    if (err != hipSuccess) {
        fprintf(stderr, "%d Failed to allocate temp device memory - %s\n", __LINE__,hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Init temp array
    initTemp<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(d_temp); // 10 blocks 20 threads; each thread has range of 256
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "%d Failed to launch temp init -  %s\n",__LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "%d Failed to copy data from host to device - %s\n",__LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //Calculate the histogram for each thread on ints segment              75000*int| 51200*int| 75000| 256
    fillTemp<<<NUM_OF_BLOCKS,NUM_OF_THREADS>>>(d_A, d_temp, size/4, cudaOutSize); //data, tempTofill, datasize, value range
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "%d Failed to launch fillTemp kernel -  %s\n",__LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Add values from each thread segment to a single array
    unify<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(d_temp, d_out);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch unify kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cudaOut, d_out, outSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

        if (hipFree(d_temp) != hipSuccess) {
        fprintf(stderr, "Failed to free device temp - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

        if (hipFree(d_out) != hipSuccess) {
        fprintf(stderr, "Failed to free device out - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

